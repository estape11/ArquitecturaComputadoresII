
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
	Aqui se defina la funcion que se quiere integrar
*/
double funcion(double x){
	return 4/(1+x*x); // mofificar con la funcion deseada

}

__global__
void aproxIntegral( double (*f)(double), double a, double b , long n, double *result ){
    double suma, tiempoInicio, tiempoEjecucion, resultado;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if ( i < n ) *result+=(*f)(a+i*(b-a)/n);    

}

int main(){
    int N = 10;
    int nblocks = (N+255) / 256;
    double a;
    aproxIntegral<<<nblocks, 256>>>(funcion, 0, 1, N, &a);

    printf("> Result %lf", a );

    return 0;

}
