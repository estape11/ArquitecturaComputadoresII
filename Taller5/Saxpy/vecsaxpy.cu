#include <stdio.h>
#include <stdexcept>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>

void saxpySerial(int n, float alpha, float *x, float *y) {
    for ( int i = 0 ; i<n ; ++i ){
        y[i] = alpha*x[i] + y[i];

    }

}

__global__
void saxpyParallel(int n, float alpha, float *x, float *y) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if ( i<n ) y[i] = alpha*x[i] + y[i];

}

int main(int argc, char* argv[]){
    std::cout<< "> Starting "<<std::endl;
    
    float *x, *y, *d_x, *d_y;

    int N = 10;

    if (argc>1) {
        N = atoi(argv[1]);
    }
    std::cout<<"> N = "<<N<<std::endl;
    x = (float *) malloc(N*sizeof(float));
    y = (float *) malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));

    for (int i = 0 ; i<N ; i++){
        x[i]=3.0;
        y[i]=4.0;

    }
    
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    int nblocks = (N+255) / 256;
    clock_t tiempoInicio, tiempoFinal;
    double resultado;

    tiempoInicio = clock();

    saxpyParallel<<<nblocks, 256>>>(N, 2.0f, d_x, d_y);
    
    tiempoFinal = clock();

    resultado = (double) (tiempoFinal-tiempoInicio)/ CLOCKS_PER_SEC;
	printf("> Total time Paralell: %lf s\n", resultado);
    
    tiempoInicio = clock();
    saxpySerial(N, 2.0, x, y);
    tiempoFinal = clock();


    std::cout<< "> Finished <"<<std::endl;
    resultado = (double) (tiempoFinal-tiempoInicio)/ CLOCKS_PER_SEC;
    printf("> Total time Serial: %lf s\n", resultado);

    hipFree(d_x);
    hipFree(d_y);
    
    free(x);
    free(y);

    return 0;

}
