#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>

__global__
void logX(float x, int N, double *result) {
    long i = blockDim.x * blockIdx.x + threadIdx.x;
    if ( i < N ) {
        *result += pow(x, i) / factorial(i);

    }
}

int main() {
    std::cout<<"> Started "<<std::endl;
    return 0;

}

